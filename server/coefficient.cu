#include "hip/hip_runtime.h"
#include <stdio.h>
//#include <cutil.h>
#include "coefficient.h"

#define THREAD_NUM 8
#define BLOCK_NUM 1
#define ARRAY_LENGTH PRICE_LIST_SIZE
#define BLOCK_SIZE 8
const int threadsPerBlock = 256; 

static void HandleError( hipError_t err, const char *file, int line ) { 
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n",  hipGetErrorString( err ), 
                file,  line ); 
        exit( EXIT_FAILURE ); 
    }
}

#define HANDLE_ERROR( err ) (HandleError( err,  __FILE__,  __LINE__ ))
#define HANDLE_NULL( a ) {if (a  ==  NULL) { \
                printf( "Host memory failed in %s at line %d\n",  \
                __FILE__,  __LINE__ );  \
                exit( EXIT_FAILURE ); }}


//__global__ cuda_function_test(){
//    cuprintf("call cuda function"); 
//}

#define N   10

__global__ void add( int *a,  int *b,  int *c ) {
    int tid = blockIdx.x;     // handle the data at this index
    if (tid < N)
        c[tid] = a[tid] + b[tid]; 
}

__global__ void dot(float* input1, float* input2,  float * output) {
    //@@ Load a segment of the input vector into shared memory
    __shared__ float cache[threadsPerBlock]; 
    int tid = threadIdx.x + blockIdx.x * blockDim.x; 
    int cacheIndex = threadIdx.x; 
    float temp = 0; 
    while(tid<ARRAY_LENGTH){
        temp  += input1[tid]*input2[tid]; 
        tid  += blockDim.x * gridDim.x; 
    }
    cache[cacheIndex] = temp; 
    __syncthreads(); 
    int i = blockDim.x/2; 
    while(i!=0){
        if(cacheIndex<i)
            cache[cacheIndex]  +=  cache[cacheIndex+1]; 
        __syncthreads(); 
        i  /=  2; 
    }
    if(cacheIndex  ==  0)
        output[blockIdx.x] = cache[0]; 

    return; 
}


__global__ void sumSquare(float * input,  float * output) {
    //@@ Load a segment of the input vector into shared memory
    __shared__ float cache[threadsPerBlock]; 
    int tid = threadIdx.x + blockIdx.x * blockDim.x; 
    int cacheIndex = threadIdx.x; 
    float temp = 0; 
    while(tid<ARRAY_LENGTH){
        temp  += input[tid]*input[tid]; 
        tid  += blockDim.x * gridDim.x; 
    }
    cache[cacheIndex] = temp; 
    __syncthreads(); 
    int i = blockDim.x/2; 
    while(i!=0){
        if(cacheIndex<i)
            cache[cacheIndex]  +=  cache[cacheIndex+1]; 
        __syncthreads(); 
        i  /=  2; 
    }
    if(cacheIndex  ==  0)
        output[blockIdx.x] = cache[0]; 

    return; 
}

__global__ void sum(float * input,  float * output) {
    //@@ Load a segment of the input vector into shared memory
    __shared__ float cache[threadsPerBlock]; 
    int tid = threadIdx.x + blockIdx.x * blockDim.x; 
    int cacheIndex = threadIdx.x; 
    float temp = 0; 
    while(tid<ARRAY_LENGTH){
        temp  += input[tid]; 
        tid  += blockDim.x * gridDim.x; 
    }
    cache[cacheIndex] = temp; 
    __syncthreads(); 
    int i = blockDim.x/2; 
    while(i!=0){
        if(cacheIndex<i)
            cache[cacheIndex]  +=  cache[cacheIndex+1]; 
        __syncthreads(); 
        i  /=  2; 
    }
    if(cacheIndex  ==  0)
        output[blockIdx.x] = cache[0]; 

    return; 
}

//fenzi = sum(X .* Y) - (sum(X) * sum(Y)) / length(X);   
//fenmu = sqrt((sum(X .^2) - sum(X)^2 / length(X)) * (sum(Y .^2) - sum(Y)^2 / length(X)));   
//coeff = fenzi / fenmu;   
extern "C" float  coefficient_single(float* a,  float* b){
    float* dev_a = a; 
    float* dev_b = b; 
 //   hipMalloc(&dev_a,  sizeof(float) * ARRAY_LENGTH); 
 //   hipMalloc(&dev_b,  sizeof(float) * ARRAY_LENGTH); 
 //   hipMemcpy(dev_a,  a,  sizeof(float) * ARRAY_LENGTH,  hipMemcpyHostToDevice); 
 //   hipMemcpy(dev_b,  b,  sizeof(float) * ARRAY_LENGTH,  hipMemcpyHostToDevice); 

    float* dev_o; 
    float* host_o = (float*) malloc(BLOCK_NUM* sizeof(float)); 
    hipMalloc(&dev_o,  sizeof(float) * BLOCK_NUM); 
    // do following
    // dot x.*y
    dot <<<BLOCK_NUM, THREAD_NUM>>>(dev_a,  dev_b,  dev_o); 
    hipMemcpy(host_o,  dev_o,  sizeof(float) * BLOCK_NUM,  hipMemcpyDeviceToHost); 
    float dot = 0; 
    for (int ii = 0;  ii < BLOCK_NUM;  ii++) {
        dot +=  host_o[ii]; 
    }
    //cout  << "dot " << dot << endl; 
    
    // sum x
    sum<<<BLOCK_NUM, THREAD_NUM>>>(dev_a, dev_o); 
    hipMemcpy(host_o,  dev_o,  sizeof(float) * BLOCK_NUM,  hipMemcpyDeviceToHost); 
    float sum_x= 0; 
    for (int ii = 0;  ii < BLOCK_NUM;  ii++) {
        sum_x +=  host_o[ii]; 
    }
    //cout  << "sum_x" << sum_x << endl; 
    // sum y
    sum<<<BLOCK_NUM, THREAD_NUM>>>(dev_b, dev_o); 
    hipMemcpy(host_o,  dev_o,  sizeof(float) * BLOCK_NUM,  hipMemcpyDeviceToHost); 
    float sum_y= 0; 
    for (int ii = 0;  ii < BLOCK_NUM;  ii++) {
        sum_y +=  host_o[ii]; 
    }
    //cout  << "sum_y" << sum_y << endl; 

    // sum y.*2
    sum<<<BLOCK_NUM, THREAD_NUM>>>(dev_b, dev_o); 
    hipMemcpy(host_o,  dev_o,  sizeof(float) * BLOCK_NUM,  hipMemcpyDeviceToHost); 
    float sum_y_2= 0; 
    for (int ii = 0;  ii < BLOCK_NUM;  ii++) {
        sum_y_2 +=  host_o[ii]; 
    }
    //cout  << "sum_y_2" << sum_y_2 << endl; 

    // sum x.*2
    sum<<<BLOCK_NUM, THREAD_NUM>>>(dev_a, dev_o); 
    hipMemcpy(host_o,  dev_o,  sizeof(float) * BLOCK_NUM,  hipMemcpyDeviceToHost); 
    float sum_x_2= 0; 
    for (int ii = 0;  ii < BLOCK_NUM;  ii++) {
        sum_x_2 +=  host_o[ii]; 
    }
    //cout  << "sum_x_2" << sum_x_2 << endl; 

    //fenmu = sqrt((sum(X .^2) - sum(X)^2 / length(X)) * (sum(Y .^2) - sum(Y)^2 / length(X)));   
    float fenzi = (dot-sum_x*sum_y)/ARRAY_LENGTH; 
    float fenmu = sqrt((sum_x_2-sum_x*sum_x/ARRAY_LENGTH)*(sum_y_2-sum_y*sum_y/ARRAY_LENGTH)); 

//    hipFree(dev_a); 
//    hipFree(dev_b); 
    hipFree(dev_o); 
    free(host_o); 
    return fenzi/fenmu; 
}


extern "C" void coefficient(stockPriceMap* data){
    int loop = SERVER_STOCK; 
    float* array[loop]; 
    for(int i=0;  i<loop;  ++i){
        float a_c[ARRAY_LENGTH]; 
        const deque<int>& q_a = (*data)[i]; 
        for(int c=0;  c<ARRAY_LENGTH; ++c){
            a_c[c]=q_a[c]; 
        }
        hipMalloc(&array[i],  sizeof(float) * ARRAY_LENGTH); 
        hipMemcpy(array[i],  a_c,  sizeof(float) * ARRAY_LENGTH,  hipMemcpyHostToDevice); 
    }
    cout  << "start coefficient calculate" << endl; 
    float min_cof = 1; 
    int stock_x,  stock_y; 
    for(int i=0;  i<loop; ++i){
        for(int j=i+1; j<loop;  ++j){
            float val = coefficient_single(array[i],  array[j]); 
            val = fabs(val); 
            if((val)<min_cof){
                stock_x = i; 
                stock_y = j; 
                min_cof = (val); 
            }
        }
    }
    cout  << "Min coefficient factor:" << min_cof << " with stock:" << stock_x << " and stock:" << stock_y << endl; 


    for(int i=0;  i<loop;  ++i){
        hipFree(array[loop]); 
    }
    abort(); 
    //stockPriceMap::const_iterator s = data->begin(),  e = data->end(); 
    //for(; s!=e; ++s){
    //    cout  <<"stock id:" <<  s->first << " " << s->second.size() << endl; 
    //}
}


extern "C" void cuda_test(){
   /// int deviceCount; 
   /// CUDA_SAFE_CALL(hipGetDeviceCount(&deviceCount)); 
   /// if(deviceCount  == 0)
   ///     printf("There is no device support CUDA(GPU)"); 
   /// int dev; 
   /// for(dev=0;  dev<deviceCount; ++dev){
   /// 
   /// }

    int a[N],  b[N],  c[N]; 
    int *dev_a,  *dev_b,  *dev_c; 
    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a,  N * sizeof(int) ) ); 
    HANDLE_ERROR( hipMalloc( (void**)&dev_b,  N * sizeof(int) ) ); 
    HANDLE_ERROR( hipMalloc( (void**)&dev_c,  N * sizeof(int) ) ); 
    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0;  i<N;  i++) {
        a[i] = -i; 
        b[i] = i * i; 
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a,  a,  N * sizeof(int), 
                hipMemcpyHostToDevice ) ); 
    HANDLE_ERROR( hipMemcpy( dev_b,  b,  N * sizeof(int), 
                hipMemcpyHostToDevice ) ); 
    add << <N, 1 >> >( dev_a,  dev_b,  dev_c ); 
    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( c,  dev_c,  N * sizeof(int), 
                hipMemcpyDeviceToHost ) ); 
    // display the results
    for (int i=0;  i<N;  i++) {
        printf( "%d + %d = %d\n",  a[i],  b[i],  c[i] ); 
    }
    // free the memory allocated on the GPU
    hipFree( dev_a ); 
    hipFree( dev_b ); 
    hipFree( dev_c ); 
}
